#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"

extern "C" {

int cuda_set_device(int i_gpu)
{
    hipError_t err = hipSetDevice(i_gpu);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaSetDevice: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cuda_get_device_count(int *n_gpu)
{
    hipError_t err = hipGetDeviceCount(n_gpu);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaGetDeviceCount: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cuda_device_synchronize()
{
    hipError_t err = hipDeviceSynchronize();

    if (err != hipSuccess)
    {
        printf("\n Error in cudaDeviceSynchronize: %s \n", hipGetErrorString(err));
    }

    return 0;
}

int cuda_malloc(intptr_t *a, size_t size)
{
    hipError_t err = hipMalloc((void **) a, size);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaMalloc: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cuda_free(intptr_t *a)
{
    hipError_t err = hipFree(a);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaFree: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cuda_memcpy(intptr_t *dest, intptr_t *src, size_t count, int dir)
{
    hipMemcpyKind dir2;

    switch (dir)
    {
        case 0:
            dir2 = hipMemcpyHostToDevice;
            break;
        case 1:
            dir2 = hipMemcpyDeviceToHost;
            break;
        case 2:
            dir2 = hipMemcpyDeviceToDevice;
            break;
    }

    hipError_t err = hipMemcpy(dest, src, count, dir2);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaMemcpy: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cublas_create(intptr_t *cublas_handle)
{
    *cublas_handle = (intptr_t) malloc(sizeof(hipblasHandle_t));
    hipblasStatus_t status = hipblasCreate((hipblasHandle_t*) *cublas_handle);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("Error in cublasCreate\n");
        exit(1);
    }

    return 0;
}

int cublas_destroy(intptr_t *cublas_handle)
{
    hipblasStatus_t status = hipblasDestroy(*((hipblasHandle_t*) *cublas_handle));
    *cublas_handle = (intptr_t) NULL;

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("Error in cublasDestory\n");
        exit(1);
    }

    return 0;
}

int cublas_set_stream(intptr_t *cublas_handle, intptr_t *hip_stream)
{
    hipblasStatus_t status = hipblasSetStream(*((hipblasHandle_t*) *cublas_handle), *((hipStream_t*) *hip_stream));

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("Error in cublasSetStream\n");
        exit(1);
    }

    return 0;
}

void cublas_dgemm(intptr_t handle, char transa, char transb, int m, int n, int k, double alpha, const double *A, int lda, const double *B, int ldb, double beta, double *C, int ldc)
{
    hipblasDgemm(*((hipblasHandle_t*)handle), HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);
}

}
